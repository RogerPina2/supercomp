#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iostream>
#include <chrono>

int main() {
    thrust::host_vector<double> host;
    // thrust::sequence(host.begin(), host.end());

    while (std::cin.good()) {
        double t;
        std::cin >> t;
        host.push_back(t);
    }
    
    thrust::device_vector<double> dev(host);
    
    // 1. O preço médio das ações nos últimos 10 anos.
    double media_gpu = thrust::reduce(dev.begin(), dev.end(), 0, thrust::plus<double>()) / dev.size();

    // 2. O preço médio das ações no último ano (365 anos atrás).
    double media_gpu_ua = thrust::reduce(dev.end()-365, dev.end(), 0, thrust::plus<double>()) / dev.size();

    // 3. O maior e o menor preço da sequência inteira e do último ano.
    double max = thrust::reduce(dev.begin(), dev.end(), 0, thrust::maximum<double>());
    double min = thrust::reduce(dev.begin(), dev.end(), 0, thrust::minimum<double>());
    double max_ua = thrust::reduce(dev.end() - 365, dev.end(), 0, thrust::minimum<double>());
    double min_ua = thrust::reduce(dev.end() - 365, dev.end(), 0, thrust::minimum<double>());

    std::cout << dev[0] << "\n";
}

